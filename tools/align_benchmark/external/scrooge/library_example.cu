#include <string>
#include <vector>
#include <iostream>

#include "genasm_cpu.hpp"
#include "genasm_gpu.hpp"
#include "util.hpp"

using namespace std;

void cpu_string_pairs_example(){
    vector<string> texts = {"ACGTACGT"};
    vector<string> queries = {"ACGTACG"};

    int threads = 1;
    vector<Alignment_t> alignments = genasm_cpu::align_all(texts, queries, threads);

    for(Alignment_t &aln : alignments){
        cout << "edit_distance:" << aln.edit_distance << " ";
        cout << "cigar:" << aln.cigar << endl;
    }
}

void gpu_string_pairs_example(){
    vector<string> texts = {"ACGTACGT"};
    vector<string> queries = {"ACGTACG"};

    vector<Alignment_t> alignments = genasm_gpu::align_all(texts, queries);

    for(Alignment_t &aln : alignments){
        cout << "edit_distance:" << aln.edit_distance << " ";
        cout << "cigar:" << aln.cigar << endl;
    }
}

void cpu_mapping_example(){
    Genome_t reference;
    reference.content = "ACGTACGT";

    //candidate locations can be anywhere in the reference genome
    //each read can have its own set of multiple candidate locations
    //here we use only a single single set with a single candidate location at the start of the reference genome, for demonstration purposes
    CandidateLocation_t ref_begin;
    ref_begin.start_in_reference = 0; //first character index of the candidate location
    ref_begin.strand = true; //forward strand
    vector<CandidateLocation_t> ref_begin_vec(1, ref_begin);

    Read_t read;
    read.description = "example_read_id";
    read.content = "ACGTACG";
    read.locations = ref_begin_vec;
    vector<Read_t> reads(1, read);

    int threads = 1;
    vector<Alignment_t> alignments = genasm_cpu::align_all(reference, reads, threads);

    for(Alignment_t &aln : alignments){
        cout << "edit_distance:" << aln.edit_distance << " ";
        cout << "cigar:" << aln.cigar << endl;
    }
}

void gpu_mapping_example(){
    Genome_t reference;
    reference.content = "ACGTACGT";

    //candidate locations can be anywhere in the reference genome
    //each read can have its own set of multiple candidate locations
    //here we use only a single single set with a single candidate location at the start of the reference genome, for demonstration purposes
    CandidateLocation_t ref_begin;
    ref_begin.start_in_reference = 0; //first character index of the candidate location
    ref_begin.strand = true; //forward strand
    vector<CandidateLocation_t> ref_begin_vec(1, ref_begin);

    
    Read_t read;
    read.description = "example_read_id";
    read.content = "ACGTACG";
    read.locations = ref_begin_vec;
    vector<Read_t> reads(1, read);

    vector<Alignment_t> alignments = genasm_gpu::align_all(reference, reads);

    for(Alignment_t &aln : alignments){
        cout << "edit_distance:" << aln.edit_distance << " ";
        cout << "cigar:" << aln.cigar << endl;
    }
}

int main(int argc, char *argv[]){
    genasm_cpu::enabled_algorithm_log = false;
    genasm_gpu::enabled_algorithm_log = false;
    
    cpu_string_pairs_example();
    gpu_string_pairs_example();
    cpu_mapping_example();
    gpu_mapping_example();
}
