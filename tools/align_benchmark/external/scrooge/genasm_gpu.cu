#include "hip/hip_runtime.h"
#ifndef CLI_KNOBS
/**********************************************************************************************/
/*************************************** [CONFIG KNOBS] ***************************************/
/**********************************************************************************************/

    //algorithm parameters
    #define W 64
    #define K 64
    #define O 33

    //optimization toggles, comment out to disable any given optimization
    #define STORE_ENTRIES_NOT_EDGES
    #define DISCARD_ENTRIES_NOT_USED_BY_TRACEBACK
    //#define EARLY_TERMINATION

    //implementation parameters
    #define THREAD_BLOCKS_PER_SM 20
    #define CIGAR_SUBLIST_SIZE 64

    //uncomment the desired memory type for the DP table R
    #define DP_MEMORY_SHARED
    //#define DP_MEMORY_GLOBAL

    //percent of L1/smem/txcache reserved for shared memory
    //comment out to let CUDA determine it automatically
    //devies of capability 8.6 support 0, 8, 16, 32, 64 or 100kiB per SM
    //devices of capability 8.0 support 0, 8, 16, 32, 64, 100, 132 or 164kiB per SM
    #define SMEM_CARVEOUT_PERCENT 100

    //#define DEBUG //uncomment to enable asserts in kernel
    //#define DEBUG_OUTPUT //uncomment to enable error messages in kernel, requires DEBUG

/**********************************************************************************************/
/************************************* [END CONFIG KNOBS] *************************************/
/**********************************************************************************************/
#else
    #define W CLI_W
    #define K CLI_K
    #define O CLI_O
    #define THREAD_BLOCKS_PER_SM CLI_THREAD_BLOCKS_PER_SM
    #define CIGAR_SUBLIST_SIZE CLI_CIGAR_SUBLIST_SIZE

    #ifdef CLI_STORE_ENTRIES_NOT_EDGES
        #define STORE_ENTRIES_NOT_EDGES
    #endif
    #ifdef CLI_DISCARD_ENTRIES_NOT_USED_BY_TRACEBACK
        #define DISCARD_ENTRIES_NOT_USED_BY_TRACEBACK
    #endif
    #ifdef CLI_EARLY_TERMINATION
        #define EARLY_TERMINATION 
    #endif

    #ifdef CLI_DP_MEMORY_SHARED
        #define DP_MEMORY_SHARED
    #endif
    #ifdef CLI_DP_MEMORY_GLOBAL
        #define DP_MEMORY_GLOBAL
    #endif

    #ifdef CLI_SMEM_CARVEOUT_PERCENT
        #define SMEM_CARVEOUT_PERCENT CLI_SMEM_CARVEOUT_PERCENT
    #endif
#endif

/* internal macros */

#define GPU_ID 0
#define THREAD_BLOCKS smCount(GPU_ID)*THREAD_BLOCKS_PER_SM

//warp and thread numbers for a single block
#define THREADS W
#define WARPS ((THREADS+31)/32)
#define ALL_THREADS 0xFFFFFFFF

//maximum number of text and pattern characters to trace back per window
#define TB_LIMIT (W-O)
//number of bits in bitvector and halfbitvector needed for traceback
#define TB_BITS min(W-O+1, m)
//number of bits in bitvector not needed for traceback
#define NON_TB_BITS (m - (TB_BITS))
//index into traceback bitvector where 0==MSB, m-1==LSB, corresponding to the pattern indices
#ifdef DISCARD_ENTRIES_NOT_USED_BY_TRACEBACK
    #define TB_BIT(J) ((TB_BITS) - 1 - (J))
#else
    #define TB_BIT(J) (m - 1 - (J))
#endif

//size of R
#ifdef STORE_ENTRIES_NOT_EDGES
    #define BITVECTORS_PER_ELEMENT 1
#else
    #define BITVECTORS_PER_ELEMENT 3
#endif
#ifdef DISCARD_ENTRIES_NOT_USED_BY_TRACEBACK
    #define COLUMNS (W-O+1)
#else
    #define COLUMNS (W+1)
#endif
#define ROWS (K+1)
#define R_BITVECTORS (COLUMNS * ROWS * BITVECTORS_PER_ELEMENT)

//indexing into R
#ifdef STORE_ENTRIES_NOT_EDGES
    #define IDX(I, D) (ROWS*(I) + (D))
#else
    #define MAT 0
    #define INS 1
    #define DEL 2
    #define IDX(I, D, EDIT_TYPE) (3*(ROWS*(I) + (D)) + (EDIT_TYPE))
#endif

//codes used in twobit representation
#define A 0x00
#define C 0x01
#define G 0x02
#define T 0x03

#include "util.hpp"
#include "cuda_util.hpp"
#include "genasm_gpu.hpp"

#include <iostream>
#include <math.h>
#include <fstream>
#include <string>
#include <assert.h>

using namespace std;

bool genasm_gpu::enabled_algorithm_log = true;

#define CUDA_LIST cigar_list
#define CUDA_LIST_SUBLIST_SIZE CIGAR_SUBLIST_SIZE
#define CUDA_LIST_CONTAINED_TYPE CigarEntry_t
#include "cuda_list.hpp"

#define BITVECTOR_NS genasm_gpu
#define BITVECTOR bitvector
#define BITVECTOR_BITS W
#include "bitvector.hpp"

#ifdef DISCARD_ENTRIES_NOT_USED_BY_TRACEBACK
    #define BITVECTOR_NS genasm_gpu
    #define BITVECTOR halfbitvector
    #if O > 0
        #define BITVECTOR_BITS (TB_LIMIT+1)
    #else
        #define BITVECTOR_BITS W
    #endif
    #include "bitvector.hpp"
#else
    namespace genasm_gpu {
        typedef bitvector halfbitvector;
    }
#endif

namespace genasm_gpu {
    typedef struct PM {
        bitvector masks[4];
    } PM_t;

    typedef struct TwoBitArray {
        const char* __restrict__ base;
        unsigned long long offset;  //offset as #ACGT characters
                                    //necessary since *base is only at 4-ACGT granularity
        unsigned long long size; //length as #ACGT characters
    } TwoBitArray_t;

    struct AlignmentRes {
        long long edit_distance;
        cigar_list cigar;
    };

    __device__ char twobit_at(unsigned int i, TwoBitArray_t twobit){
        unsigned int byte_idx = (i+twobit.offset)>>2;
        char subbyte_idx = (i+twobit.offset)%4;

        char quad_code = twobit.base[byte_idx];
        char shifted_quad_code = quad_code >> (6 - (subbyte_idx<<1));
        return shifted_quad_code & 0x3;
    }

    __host__ __device__ TwoBitArray_t twobit_add(TwoBitArray_t twobit, int x){
        TwoBitArray_t res;
        res.base = twobit.base;
        res.offset = twobit.offset + x;
        res.size = twobit.size - x;
        return res;
    }

    __device__ void print_twobit_as_ascii(long long length, TwoBitArray_t twobit){
        char *res = (char *)malloc(length + 1);
        for(int i = 0; i < length; i++){
            char code = twobit_at(i, twobit);
            if(code == A) res[i] = 'A';
            if(code == C) res[i] = 'C';
            if(code == G) res[i] = 'G';
            if(code == T) res[i] = 'T';
        }
        res[length] = '\0';
        printf("%s\n", res);
        free(res);
    }

    __device__ PM_t generatePatternBitmaskACGT(int m, TwoBitArray_t pattern){
        int bit_idx = threadIdx.x;
        int i = m - 1 - bit_idx;
        int warp_bit_idx = bit_idx%32;

        uint32_t warp_masks[4];
        warp_masks[A] = 0xFFFFFFFF;
        warp_masks[C] = 0xFFFFFFFF;
        warp_masks[G] = 0xFFFFFFFF;
        warp_masks[T] = 0xFFFFFFFF;

        if(i >= 0){
            char code = twobit_at(i, pattern);
            warp_masks[code] = ~(1u<<warp_bit_idx);
        }

        for (int offset = 16; offset > 0; offset /= 2){
            int t = threadIdx.x;
            bool is_recipient = t%32<offset;
            bool transmitter_valid = t+offset <blockDim.x;
            bool do_shuffle = transmitter_valid || !is_recipient;
            uint32_t threads_mask = __ballot_sync(ALL_THREADS, do_shuffle);
            if(!do_shuffle)
                continue;

            warp_masks[A] &= __shfl_down_sync(threads_mask, warp_masks[A], offset);
            warp_masks[C] &= __shfl_down_sync(threads_mask, warp_masks[C], offset);
            warp_masks[G] &= __shfl_down_sync(threads_mask, warp_masks[G], offset);
            warp_masks[T] &= __shfl_down_sync(threads_mask, warp_masks[T], offset);
        }

        __shared__ PM_t pm;
        if(threadIdx.x == 0){
            pm.masks[A] = bitvector::zeros();
            pm.masks[C] = bitvector::zeros();
            pm.masks[G] = bitvector::zeros();
            pm.masks[T] = bitvector::zeros();
        }
        __syncthreads();

        if(bit_idx % 32 == 0){
            pm.masks[A].insert_bits_threadsafe(bit_idx, warp_masks[A]);
            pm.masks[C].insert_bits_threadsafe(bit_idx, warp_masks[C]);
            pm.masks[G].insert_bits_threadsafe(bit_idx, warp_masks[G]);
            pm.masks[T].insert_bits_threadsafe(bit_idx, warp_masks[T]);
        }

        __syncthreads();

        return pm;
    }

    __device__ halfbitvector extract_tb_bitvector(int m, bitvector b){
        halfbitvector res = halfbitvector::zeros();
        for(int next_bit_offset = 0; next_bit_offset < TB_BITS; next_bit_offset+=32){
            int next_bit = NON_TB_BITS + next_bit_offset;
            uint32_t tmp = b.extract_bits(next_bit);
            res.insert_bits(next_bit_offset, tmp);
        }
        return res;
    }

    __device__ void print_state(int n, TwoBitArray_t text, int m, TwoBitArray_t pattern, int k, halfbitvector *R){
        for(int it = min((W-O), m); it >= 0; it--){
            #ifdef STORE_ENTRIES_NOT_EDGES
                printf("text_iteration = %d\n", it);
                for(int dt = 0; dt <= k; dt++){
                    R[IDX(it, dt)].print();
                }
            #else
                printf("text_iteration = %d insertion\n", it);
                for(int dt = 0; dt <= k; dt++){
                    R[IDX(it, dt, INS)].print();
                }
                printf("text_iteration = %d deletion\n", it);
                for(int dt = 0; dt <= k; dt++){
                    R[IDX(it, dt, DEL)].print();
                }
                printf("text_iteration = %d match\n", it);
                for(int dt = 0; dt <= k; dt++){
                    R[IDX(it, dt, MAT)].print();
                }
            #endif
        }

        printf("text   =");
        print_twobit_as_ascii(n, text);
        printf("pattern=");
        print_twobit_as_ascii(m, pattern);
    }

    __device__ int genasm_dc(int n, TwoBitArray_t text, int m, TwoBitArray_t pattern, int k, halfbitvector* __restrict__ R
    #ifdef DEBUG
        , size_t read_number
    #endif
    ){
        PM_t pm = generatePatternBitmaskACGT(m, pattern);

        //if n is within the area TB may reach we need to initialize it as well
        #ifdef STORE_ENTRIES_NOT_EDGES
            #ifdef DISCARD_ENTRIES_NOT_USED_BY_TRACEBACK
                if(n < (W-O+1)){
                    if(threadIdx.x == 0){
                        R[IDX(n, 0)] = halfbitvector::ones();
                    }
                    
                    int d = threadIdx.x+1;
                    R[IDX(n, d)] = halfbitvector::ones() << max(0, d - NON_TB_BITS);
                }
            #else
                {
                if(threadIdx.x == 0){
                    R[IDX(n, 0)] = halfbitvector::ones();
                }
                int d = threadIdx.x+1;
                R[IDX(n, d)] = halfbitvector::ones() << d;
                }
            #endif
        #endif

        int i = n - threadIdx.x - 1;

        char curChar;
        bitvector curPm;
        if(i >= 0){
            curChar = twobit_at(i, text);
            curPm = pm.masks[curChar];
        }

        int window_edit_distance;
        window_edit_distance = m;

        #ifdef EARLY_TERMINATION
            __shared__ bool early_terminate;
            if(threadIdx.x == 0){
                early_terminate = false;
            }
        #endif

        bitvector top, left, topleft, center, diag;
        bitvector mat, sub, ins, del;
        mat = bitvector::ones();
        sub = bitvector::ones();
        ins = bitvector::ones();
        del = bitvector::ones();
        int d = 0;
        for(int cycle = 0; cycle < n+k; cycle++){
            __syncthreads();
            #ifdef EARLY_TERMINATION
                if(early_terminate) break;
            #endif
            diag = bitvector::shuffle_up(center);

            if(cycle < threadIdx.x) continue;
            if(i < 0) continue;
            if(d > k) continue;

            topleft = left;
            if(i == n-1){
                left = bitvector::ones() << d;
            }
            else{
                left = diag;
            }
            top = center;

            if(d == 0){
                mat = (left << 1) | curPm;
                center = mat;
            }
            else{
                del = topleft;
                sub = topleft << 1;
                ins = top << 1;
                mat = (left << 1) | curPm;
                center = del & sub & ins & mat;
            }

            #ifdef DISCARD_ENTRIES_NOT_USED_BY_TRACEBACK
                if(i < (W-O+1)){
                    #ifdef STORE_ENTRIES_NOT_EDGES
                        R[IDX(i, d)] = extract_tb_bitvector(m, center);
                    #else
                        R[IDX(i, d, MAT)] = extract_tb_bitvector(m, mat);
                        R[IDX(i, d, INS)] = extract_tb_bitvector(m, ins);
                        R[IDX(i, d, DEL)] = extract_tb_bitvector(m, del);
                    #endif
                }
            #else
                #ifdef STORE_ENTRIES_NOT_EDGES
                    R[IDX(i, d)] = center;
                #else
                    R[IDX(i, d, MAT)] = mat;
                    R[IDX(i, d, INS)] = ins;
                    R[IDX(i, d, DEL)] = del;
                #endif
            #endif

            if(center.has_zero_at(m-1)){
                window_edit_distance = min(d, window_edit_distance);
                #ifdef EARLY_TERMINATION
                    if(i == 0){
                        early_terminate = true;
                    }
                #endif
            }

            d++;
        }
        __shared__ int shared_window_edit_distance;
        if(i == 0){
            shared_window_edit_distance = window_edit_distance;
        }
        __syncthreads();
        return shared_window_edit_distance;
    }

    __device__ void genasm_tb(int n, TwoBitArray_t text, int m, TwoBitArray_t pattern, int k, halfbitvector* __restrict__ R, int window_edit_distance, int *text_consumed, int *pattern_consumed, int *edits_used, cigar_list *cigar_list
    #ifdef DEBUG
        , size_t read_number
    #endif
    ){
        if(threadIdx.x != 0) return;

        int i = 0;
        int j = 0;
        int d = window_edit_distance;

        char current_edit_type = ' ';
        uint8_t current_edit_count = 0;

        while(j < m){ //trailing deletes are ignored, so we can terminate as soon as pattern is fully matches
            //early termination when windowing
            if(i >= TB_LIMIT) break;
            if(j >= TB_LIMIT) break;

            bool i_limit = i >= n;
            bool d_limit = d == 0;

            bool can_sub, can_ins, can_del;
            #ifdef DEBUG
                bool can_mat;
            #endif
            if(j < m-1){
                #ifdef STORE_ENTRIES_NOT_EDGES
                    can_ins = !d_limit &&             R[IDX(i  , d-1)].has_zero_at(TB_BIT(j+1));
                    can_del = !d_limit && !i_limit && R[IDX(i+1, d-1)].has_zero_at(TB_BIT(j  ));
                    can_sub = !d_limit && !i_limit && R[IDX(i+1, d-1)].has_zero_at(TB_BIT(j+1));
                    #ifdef DEBUG
                        can_mat =         !i_limit && R[IDX(i+1, d  )].has_zero_at(TB_BIT(j+1));
                    #endif
                #else
                    can_ins = R[IDX(i, d, INS)].has_zero_at(TB_BIT(j));
                    can_del = R[IDX(i, d, DEL)].has_zero_at(TB_BIT(j));
                    can_sub = R[IDX(i, d, DEL)].has_zero_at(TB_BIT(j+1));
                    #ifdef DEBUG
                        can_mat = R[IDX(i, d, MAT)].has_zero_at(TB_BIT(j));
                    #endif
                #endif
            }
            else{
                can_ins = !d_limit;
                can_del = false;
                can_sub = !d_limit && !i_limit;
                #ifdef DEBUG
                    can_mat = d==0;
                #endif
            }

            char edit_type;
            if(can_ins){
                j += 1;
                d -= 1;
                edit_type = 'I';
            }
            else if(can_del){
                i += 1;
                d -= 1;
                edit_type = 'D';
            }
            else if(can_sub){
                i += 1;
                j += 1;
                d -= 1;
                edit_type = 'X';
            }
            #ifdef DEBUG
                else if(can_mat){
            #else
                else{
            #endif
                i += 1;
                j += 1;
                edit_type = '=';
            }
            #ifdef DEBUG
                else if(threadIdx.x == 0){
                    #ifdef DEBUG_OUTPUT
                        printf("genasm_tb got stuck in dead-end at read number %lld\n", read_number);
                        printf("i = %d\n", i);
                        printf("j = %d\n", j);
                        printf("d = %d\n", d);
                        printf("m = %d\n", m);
                        printf("n = %d\n", n);
                        printf("k = %d\n", k);
                        printf("\n");
                        print_state(n, text, m, pattern, k, R);
                    #endif
                    assert(false);
                }
            #endif

            if(edit_type != current_edit_type){
                if(current_edit_count > 0){
                    cigar_list->pushBack({current_edit_count, current_edit_type});
                }
                current_edit_type = edit_type;
                current_edit_count = 1;
            }
            else{
                current_edit_count++;
            }
        }

        if(current_edit_count > 0){
            cigar_list->pushBack({current_edit_count, current_edit_type});
        }
        *text_consumed = i;
        *pattern_consumed = j;
        *edits_used = window_edit_distance - d;
    }

    __device__ AlignmentRes genasm(TwoBitArray_t reference, TwoBitArray_t read, halfbitvector *R
    #ifdef DEBUG
        , size_t read_number
    #endif
    ){
        __shared__ AlignmentRes res;
        if(threadIdx.x == 0){
            res.cigar.init();
            res.edit_distance = 0;
        }

        size_t reference_idx = 0;
        size_t read_idx = 0;

        while(read_idx < read.size){
            int n = min((unsigned long long)W, reference.size - reference_idx);
            int m = min((unsigned long long)W, read.size - read_idx);
            TwoBitArray_t text = twobit_add(reference, reference_idx);
            TwoBitArray_t pattern = twobit_add(read, read_idx);

            #ifdef DEBUG
                int window_edit_distance = genasm_dc(n, text, m, pattern, K, R, read_number);
            #else
                int window_edit_distance = genasm_dc(n, text, m, pattern, K, R);
            #endif

            __shared__ int text_consumed, pattern_consumed, edits_used;
            #ifdef DEBUG
                genasm_tb(n, text, m, pattern, K, R, window_edit_distance, &text_consumed, &pattern_consumed, &edits_used, &res.cigar, read_number);
            #else
                genasm_tb(n, text, m, pattern, K, R, window_edit_distance, &text_consumed, &pattern_consumed, &edits_used, &res.cigar);
            #endif
            __syncthreads();

            if(threadIdx.x == 0){
                res.edit_distance += edits_used;
            }
            reference_idx += text_consumed;
            read_idx += pattern_consumed;
        }

        __syncthreads();
        return res;
    }

    __managed__ size_t next_pair_index;
    __global__ void genasm_kernel(size_t pairs_count, TwoBitArray_t *references, TwoBitArray_t *reads, AlignmentRes *results){ 
        #ifdef DP_MEMORY_SHARED
            extern __shared__ halfbitvector R[];
        #endif
        #ifdef DP_MEMORY_GLOBAL
            __shared__ halfbitvector *R;
            if(threadIdx.x == 0){
                R = (halfbitvector*)malloc(sizeof(halfbitvector)* R_BITVECTORS);
                #ifdef DEBUG
                    if(R == NULL){
                        #ifdef DEBUG_OUTPUT
                            printf("failed to allocate R for %d bitvectors of %lld bytes each (%lld total)\n", R_BITVECTORS, sizeof(halfbitvector), sizeof(halfbitvector)* R_BITVECTORS);
                        #endif
                        assert(false);
                    }
                #endif
            }
        #endif
        
        __shared__ size_t i;
        if(threadIdx.x == 0){
            i = atomicAdd((unsigned long long *)&next_pair_index, 1);
        }
        __syncthreads();

        while(i < pairs_count){
            TwoBitArray_t ref = references[i];
            TwoBitArray_t read = reads[i];

            #ifdef DEBUG
                results[i] = genasm(ref, read, R, i);
            #else
                results[i] = genasm(ref, read, R);
            #endif

            if(threadIdx.x == 0){
                i = atomicAdd((unsigned long long *)&next_pair_index, 1);
            }
            __syncthreads();
        }

        #ifdef DP_MEMORY_GLOBAL
            if(threadIdx.x==0){
                free(R);
            }
        #endif
    }

    __device__ char ascii_to_twobit_code(char ascii){
        if(ascii == 'a' || ascii == 'A') return 0x00;
        if(ascii == 'c' || ascii == 'C') return 0x01;
        if(ascii == 'g' || ascii == 'G') return 0x02;
        if(ascii == 't' || ascii == 'T') return 0x03;
        assert(false); //invalid character
        return 0xFF;
    }

    __device__ void ascii_to_twobit_string(long long length, char *ascii, char* twobit){
        long long quad_offset = threadIdx.x;
        int quad_stride = blockDim.x;

        long long quad_idx = quad_offset;
        for(quad_idx = quad_offset; quad_idx*4+3 < length; quad_idx += quad_stride){
            long long i = quad_idx * 4;

            char c0 = ascii[i+0];
            char c1 = ascii[i+1];
            char c2 = ascii[i+2];
            char c3 = ascii[i+3];
            
            char b0 = ascii_to_twobit_code(c0);        
            char b1 = ascii_to_twobit_code(c1);        
            char b2 = ascii_to_twobit_code(c2);        
            char b3 = ascii_to_twobit_code(c3);

            char b01 = (b0 << 2) | b1;
            char b23 = (b2 << 2) | b3;
            char res = (b01 << 4) | b23;

            twobit[quad_idx] = res;
        }

        if(quad_idx*4 < length){
            char res = 0x00;
            for(long long i = 0; quad_idx*4 + i < length; i++){
                char c = ascii[quad_idx*4 + i];
                res |= ascii_to_twobit_code(c) << (6 - 2*i);
            }
            twobit[quad_idx] = res;
        }
    }

    __global__ void ascii_to_twobit_strings(int count, long long *string_lengths, char **ascii_strings, char **twobit_strings){
        int offset = blockIdx.x;
        int stride = gridDim.x;
        for(int i = offset; i < count; i+=stride){
            ascii_to_twobit_string(string_lengths[i], ascii_strings[i], twobit_strings[i]);
        }
    }

    __global__ void single_ascii_to_twobit_string(long long length, char *ascii, char *twobit){
        ascii_to_twobit_string(length, ascii, twobit);
    }

    /*
     * Given a reference genome
     * convert it to twobit representation and provide list of pointers, one for each candidate location in reads
     * out_twobit_blob is the allocated storage, should later be freed with hipFree()
     */
    void twobit_reference(Genome_t &reference, vector<Read_t> &reads, char **out_twobit_blob, TwoBitArray_t **out_twobit_arrays){
        size_t ref_len = reference.content.size();
        size_t twobit_ref_len = (ref_len+3)/4;

        size_t total_pairs = 0;
        for(size_t i = 0; i < reads.size(); i++){
            total_pairs += reads[i].locations.size();
        }

        char *reference_cudamem,
            *twobit_blob;
        TwoBitArray_t *twobit_arrays;

        CUDACHK(hipMallocManaged(&twobit_arrays, sizeof(TwoBitArray_t) * total_pairs));
        CUDACHK(hipMallocManaged(&reference_cudamem, ref_len));
        CUDACHK(hipMallocManaged(&twobit_blob, twobit_ref_len));

        memcpy(reference_cudamem, reference.content.c_str(), ref_len);

        single_ascii_to_twobit_string<<<THREAD_BLOCKS, 32>>>(ref_len, reference_cudamem, twobit_blob);
        CUDACHK(hipPeekAtLastError());
        CUDACHK(hipDeviceSynchronize());

        // V here
        size_t pair_idx = 0;
        for(size_t i = 0; i < reads.size(); i++){
            for(CandidateLocation_t &location : reads[i].locations){
                TwoBitArray_t ref;
                ref.base = twobit_blob;
                ref.offset = location.start_in_reference;
                ref.size = reference.content.size() - location.start_in_reference;
                twobit_arrays[pair_idx] = ref;
                pair_idx++;
            }
        }
        // A 

        *out_twobit_blob = twobit_blob;
        *out_twobit_arrays = twobit_arrays;

        CUDACHK(hipFree(reference_cudamem));

        //hint reads data as "readMostly"
        CUDACHK(hipMemAdvise(twobit_blob, twobit_ref_len, hipMemAdviseSetReadMostly, GPU_ID));
        CUDACHK(hipMemAdvise(twobit_arrays, sizeof(TwoBitArray_t) * total_pairs, hipMemAdviseSetReadMostly, GPU_ID));

        //explicitly prefetch reads data to GPU if possible on the current device
        if(canPrefetch(GPU_ID)){
            CUDACHK(hipMemPrefetchAsync(twobit_blob, twobit_ref_len, GPU_ID));
            CUDACHK(hipMemPrefetchAsync(twobit_arrays, sizeof(TwoBitArray_t) * total_pairs, GPU_ID));
        }
    }

    /*
     * Given a list of reads with candidate locations,
     * convert them to twobit representation and provide list of pointers, one for each candidate location
     * out_twobit_blob is the allocated storage, should later be freed with hipFree()
     */
    void twobit_reads(vector<Read_t> &reads, char **out_twobit_blob, TwoBitArray_t **out_twobit_arrays){
        char *reads_cudamem,
            *twobit_blob;
        TwoBitArray_t *twobit_arrays;

        //count total reads characters and number of alignment pairs
        size_t total_reads_size = 0;
        size_t total_pairs = 0;
        for(size_t i = 0; i < reads.size(); i++){
            total_reads_size += reads[i].content.size();
            total_pairs += reads[i].locations.size();
        }
        size_t total_reads_size_twobit = (total_reads_size+3)/4;

        //allocate input and output for conversion kernel
        CUDACHK(hipMallocManaged(&twobit_arrays, sizeof(TwoBitArray_t) * total_pairs));
        CUDACHK(hipMallocManaged(&reads_cudamem, total_reads_size));
        CUDACHK(hipMallocManaged(&twobit_blob, total_reads_size_twobit));

        //concatenate reads into kernel input
        size_t next_read_offset = 0;
        for(size_t i = 0; i < reads.size(); i++){
            memcpy(reads_cudamem + next_read_offset, reads[i].content.c_str(), reads[i].content.size());
            next_read_offset += reads[i].content.size();
        }

        //convert from byte-per-bp to twobit-per-bp
        single_ascii_to_twobit_string<<<THREAD_BLOCKS, 32>>>(total_reads_size, reads_cudamem, twobit_blob);
        CUDACHK(hipPeekAtLastError());
        CUDACHK(hipDeviceSynchronize());

        //initialize pointers into twobitarray, one for each alignment pair
        size_t pair_idx = 0;
        next_read_offset = 0;
        for(size_t i = 0; i < reads.size(); i++){
            TwoBitArray_t read;
            read.base = twobit_blob;
            read.offset = next_read_offset;
            read.size = reads[i].content.size();

            for(CandidateLocation_t &location : reads[i].locations){
                twobit_arrays[pair_idx] = read;
                pair_idx++;
            }

            next_read_offset += reads[i].content.size();
        }

        //write to out parameters
        *out_twobit_blob = twobit_blob;
        *out_twobit_arrays = twobit_arrays;

        //no longer need the input to conversion kernel
        CUDACHK(hipFree(reads_cudamem));

        //hint reads data as "readMostly"
        CUDACHK(hipMemAdvise(twobit_blob, total_reads_size_twobit, hipMemAdviseSetReadMostly, GPU_ID));
        CUDACHK(hipMemAdvise(twobit_arrays, sizeof(TwoBitArray_t) * total_pairs, hipMemAdviseSetReadMostly, GPU_ID));

        //explicitly prefetch reads data to GPU if possible on the current device
        if(canPrefetch(GPU_ID)){
            CUDACHK(hipMemPrefetchAsync(twobit_blob, total_reads_size_twobit, GPU_ID));
            CUDACHK(hipMemPrefetchAsync(twobit_arrays, sizeof(TwoBitArray_t) * total_pairs, GPU_ID));
        }
    }

    /*
     * Given a list of strings,
     * convert them to twobit representation and provide list of pointers, one for each string
     * out_twobit_blob is the allocated storage, should later be freed with hipFree()
     */
    void twobit_strings(vector<string> strings, char **out_twobit_blob, TwoBitArray_t **out_twobit_arrays){
        char *strings_cudamem,
            *twobit_blob;
        TwoBitArray_t *twobit_arrays;

        //count total reads characters and number of alignment pairs
        size_t total_strings_size = 0;
        for(size_t i = 0; i < strings.size(); i++){
            total_strings_size += strings[i].size();
        }
        size_t total_strings_size_twobit = (total_strings_size+3)/4;

        //allocate input and output for conversion kernel
        CUDACHK(hipMallocManaged(&twobit_arrays, sizeof(TwoBitArray_t) * strings.size()));
        CUDACHK(hipMallocManaged(&strings_cudamem, total_strings_size));
        CUDACHK(hipMallocManaged(&twobit_blob, total_strings_size_twobit));

        //concatenate strings into kernel input
        size_t next_string_offset = 0;
        for(size_t i = 0; i < strings.size(); i++){
            memcpy(strings_cudamem + next_string_offset, strings[i].c_str(), strings[i].size());
            next_string_offset += strings[i].size();
        }

        //convert from byte-per-bp to twobit-per-bp
        single_ascii_to_twobit_string<<<THREAD_BLOCKS, 32>>>(total_strings_size, strings_cudamem, twobit_blob);
        CUDACHK(hipPeekAtLastError());
        CUDACHK(hipDeviceSynchronize());

        //initialize pointers into twobitarray, one for each alignment pair
        next_string_offset = 0;
        for(size_t i = 0; i < strings.size(); i++){
            TwoBitArray_t tba;
            tba.base = twobit_blob;
            tba.offset = next_string_offset;
            tba.size = strings[i].size();

            twobit_arrays[i] = tba;

            next_string_offset += strings[i].size();
        }

        //write to out parameters
        *out_twobit_blob = twobit_blob;
        *out_twobit_arrays = twobit_arrays;

        //no longer need the input to conversion kernel
        CUDACHK(hipFree(strings_cudamem));

        //hint reads data as "readMostly"
        CUDACHK(hipMemAdvise(twobit_blob, total_strings_size_twobit, hipMemAdviseSetReadMostly, GPU_ID));
        CUDACHK(hipMemAdvise(twobit_arrays, sizeof(TwoBitArray_t) * strings.size(), hipMemAdviseSetReadMostly, GPU_ID));

        //explicitly prefetch reads data to GPU if possible on the current device
        if(canPrefetch(GPU_ID)){
            CUDACHK(hipMemPrefetchAsync(twobit_blob, total_strings_size_twobit, GPU_ID));
            CUDACHK(hipMemPrefetchAsync(twobit_arrays, sizeof(TwoBitArray_t) * strings.size(), GPU_ID));
        }
    }

    string cigarlist_to_cigar(cigar_list cl){
        stringstream cigar_ss;
        for(cigar_list_iterator it = cl.begin(); it != cl.end(); it++){
            cigar_ss << (int)it->edit_count;
            cigar_ss << it->edit_type;
        }
        return cigar_ss.str();
    }

    vector<Alignment_t> align_all(Genome_t &reference, vector<Read_t> &reads, long long* core_algorithm_ns){
        long long ref_len = reference.content.size();
        size_t num_pairs = 0;
        for(Read_t &read : reads){
            num_pairs += read.locations.size();
        }

        char *reference_twobit_blob,
             *reads_twobit_blob;
        TwoBitArray_t *reference_tbas,
                      *read_tbas;
        AlignmentRes *results;

        CUDACHK(hipMallocManaged(&results, num_pairs * sizeof(AlignmentRes)));

        //reserve the amount of memory at most needed for output cigars
        size_t total_cigar_sublists = 0;
        for(Read_t &read : reads){
            size_t max_edits = read.content.size()*2;
            size_t max_cigar_sublists = (max_edits+CIGAR_SUBLIST_SIZE-1)/CIGAR_SUBLIST_SIZE;
            total_cigar_sublists += max((size_t)1, max_cigar_sublists) * read.locations.size();
        }
        cigar_list::backingStorageInit(total_cigar_sublists);

        twobit_reads(reads, &reads_twobit_blob, &read_tbas);
        twobit_reference(reference, reads, &reference_twobit_blob, &reference_tbas);
        
        if(enabled_algorithm_log) cerr << "Starting Kernel..." << endl;

        size_t dp_mem_per_block = R_BITVECTORS * sizeof(halfbitvector);
        if(enabled_algorithm_log) cerr << "using " << dp_mem_per_block << "B DP memory per thread block" << endl;
        #ifdef DP_MEMORY_GLOBAL
            int safety_factor = 2; //CUDA crashes if we request the exact number of bytes
            setMallocHeapLimit(R_BITVECTORS * sizeof(halfbitvector) * THREAD_BLOCKS * safety_factor);
        #endif
        #ifdef SMEM_CARVEOUT_PERCENT
            smemCarveout(SMEM_CARVEOUT_PERCENT, (void*)genasm_kernel);
        #endif
        #ifdef DP_MEMORY_SHARED
            int smem_limit = maximizeDynamicSmem((void*)genasm_kernel, GPU_ID);
            int dp_smem = R_BITVECTORS * sizeof(halfbitvector);
            if(dp_smem > smem_limit){
                cout << "R requires " << dp_smem << "B, more than the device limit " << smem_limit << "B" << endl;
                exit(1);
            }
        #else
            int dp_smem = 0;
        #endif

        next_pair_index = 0;
        long long ns = measure_ns([&](){
            genasm_kernel<<<THREAD_BLOCKS, THREADS, dp_smem>>>(num_pairs, reference_tbas, read_tbas, results);
            CUDACHK(hipPeekAtLastError());
            CUDACHK(hipDeviceSynchronize());
        });

        if(core_algorithm_ns != NULL){
            *core_algorithm_ns = ns;
        }

        long long alignments_per_second = num_pairs * 1000000000 / ns;
        if(enabled_algorithm_log) cerr << "core algorithm ran at " << alignments_per_second << " aligns/second" << endl;

        if(enabled_algorithm_log) cerr << "Post Processing Results..." << endl;

        if(canPrefetch(hipCpuDeviceId)){
            CUDACHK(hipMemPrefetchAsync(results, num_pairs*sizeof(AlignmentRes), hipCpuDeviceId));
            cigar_list::backingStoragePrefetch(hipCpuDeviceId);
        }

        vector<Alignment_t> alignments(num_pairs);
        size_t pair_idx = 0;
        for(Read_t &read : reads){
            for(CandidateLocation_t &location : read.locations){
                alignments[pair_idx].edit_distance = results[pair_idx].edit_distance;
                alignments[pair_idx].cigar = cigarlist_to_cigar(results[pair_idx].cigar);
                pair_idx++;
            }
        }

        CUDACHK(hipFree(reference_twobit_blob));
        CUDACHK(hipFree(reads_twobit_blob));
        CUDACHK(hipFree(reference_tbas));
        CUDACHK(hipFree(read_tbas));
        CUDACHK(hipFree(results));

        cigar_list::backingStorageDestruct();

        if(enabled_algorithm_log) cerr << "pairs=" << num_pairs << " alignments=" << alignments.size() << endl;
        return alignments;
    }

    vector<Alignment_t> align_all(vector<string> &texts, vector<string> &queries, long long* core_algorithm_ns){
        size_t num_pairs = texts.size();
        assert(queries.size() == num_pairs);

        char *texts_twobit_blob,
             *queries_twobit_blob;
        TwoBitArray_t *texts_tbas,
                      *queries_tbas;
        AlignmentRes *results;

        CUDACHK(hipMallocManaged(&results, num_pairs * sizeof(AlignmentRes)));

        //reserve the amount of memory at most needed for output cigars
        size_t total_cigar_sublists = 0;
        for(string &query : queries){
            size_t max_edits = query.size()*2;
            size_t max_cigar_sublists = (max_edits+CIGAR_SUBLIST_SIZE-1)/CIGAR_SUBLIST_SIZE;
            total_cigar_sublists += max((size_t)1, max_cigar_sublists);
        }
        cigar_list::backingStorageInit(total_cigar_sublists);

        twobit_strings(queries, &queries_twobit_blob, &queries_tbas);
        twobit_strings(texts, &texts_twobit_blob, &texts_tbas);
        
        if(enabled_algorithm_log) cerr << "Starting Kernel..." << endl;

        size_t dp_mem_per_block = R_BITVECTORS * sizeof(halfbitvector);
        if(enabled_algorithm_log) cerr << "using " << dp_mem_per_block << "B DP memory per thread block" << endl;
        #ifdef DP_MEMORY_GLOBAL
            int safety_factor = 2; //CUDA crashes if we request the exact number of bytes
            setMallocHeapLimit(R_BITVECTORS * sizeof(halfbitvector) * THREAD_BLOCKS * safety_factor);
        #endif
        #ifdef SMEM_CARVEOUT_PERCENT
            smemCarveout(SMEM_CARVEOUT_PERCENT, (void*)genasm_kernel);
        #endif
        #ifdef DP_MEMORY_SHARED
            int smem_limit = maximizeDynamicSmem((void*)genasm_kernel, GPU_ID);
            int dp_smem = R_BITVECTORS * sizeof(halfbitvector);
            if(dp_smem > smem_limit){
                cout << "R requires " << dp_smem << "B, more than the device limit " << smem_limit << "B" << endl;
                exit(1);
            }
        #else
            int dp_smem = 0;
        #endif

        next_pair_index = 0;
        long long ns = measure_ns([&](){
            genasm_kernel<<<THREAD_BLOCKS, THREADS, dp_smem>>>(num_pairs, texts_tbas, queries_tbas, results);
            CUDACHK(hipPeekAtLastError());
            CUDACHK(hipDeviceSynchronize());
        });
        
        if(core_algorithm_ns != NULL){
            *core_algorithm_ns = ns;
        }

        long long alignments_per_second = num_pairs * 1000000000 / ns;
        if(enabled_algorithm_log) cerr << "core algorithm ran at " << alignments_per_second << " aligns/second" << endl;

        if(enabled_algorithm_log) cerr << "Post Processing Results..." << endl;

        if(canPrefetch(hipCpuDeviceId)){
            CUDACHK(hipMemPrefetchAsync(results, num_pairs*sizeof(AlignmentRes), hipCpuDeviceId));
            cigar_list::backingStoragePrefetch(hipCpuDeviceId);
        }

        vector<Alignment_t> alignments(num_pairs);
        for(size_t pair_idx = 0; pair_idx < num_pairs; pair_idx++){
            alignments[pair_idx].edit_distance = results[pair_idx].edit_distance;
            alignments[pair_idx].cigar = cigarlist_to_cigar(results[pair_idx].cigar);
        }

        CUDACHK(hipFree(texts_twobit_blob));
        CUDACHK(hipFree(queries_twobit_blob));
        CUDACHK(hipFree(texts_tbas));
        CUDACHK(hipFree(queries_tbas));
        CUDACHK(hipFree(results));

        cigar_list::backingStorageDestruct();

        if(enabled_algorithm_log) cerr << "pairs=" << num_pairs << " alignments=" << alignments.size() << endl;
        return alignments;
    }
}
