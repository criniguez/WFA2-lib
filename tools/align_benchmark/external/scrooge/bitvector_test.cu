
#include <iostream>
#include <iomanip>

#define BITVECTOR_BITS 65
#define BITVECTOR bv
#define BITVECTOR_ELEMENT_TYPE uint32_t
#define BITVECTOR_ELEMENT_BITS 32
#include "bitvector.hpp"

#define BITVECTOR_BITS 32
#define BITVECTOR hbv
#include "bitvector.hpp"

using namespace std;

#define TEST(CONDITION) if(!(CONDITION)) { \
    cout << "FAILED " << __func__ << " at " << __FILE__ << ":" << __LINE__ << endl; \
    passed = false; \
    }

bool shift_test(){
    bool passed = true;

    bv a = {0x89ABCDEF, 0x01234567, 0x1};
    TEST((a <<  0 == bv{0x89ABCDEF, 0x01234567, 0x1}))
    TEST((a <<  4 == bv{0x9ABCDEF0, 0x12345678, 0x0}))
    TEST((a <<  8 == bv{0xABCDEF00, 0x23456789, 0x1}))
    TEST((a << 12 == bv{0xBCDEF000, 0x3456789A, 0x0}))
    TEST((a << 16 == bv{0xCDEF0000, 0x456789AB, 0x1}))
    TEST((a << 20 == bv{0xDEF00000, 0x56789ABC, 0x0}))
    TEST((a << 24 == bv{0xEF000000, 0x6789ABCD, 0x1}))
    TEST((a << 65 == bv::zeros()))
    
    return passed;
}

bool or_test(){
    bool passed = true;

    bv a = {0xFFFF0000, 0xFFFF0000, 0x0};
    bv b = {0x0000FFFF, 0x0000FFFF, 0x1};

    bv c = {0x0F0F0F0F, 0x0F0F0F0F, 0x0};
    bv d = {0xF0F0F0F0, 0xF0F0F0F0, 0x1};

    bv e = {0x0A0A0A0A, 0x0A0A0A0A, 0x0};
    bv f = {0x05050505, 0x05050505, 0x0};

    TEST(((a | b) == bv::ones()))
    TEST(((c | d) == bv::ones()))
    TEST(((e | f) == c))
    TEST(((bv::ones() | bv::zeros()) == bv::ones()))

    return passed;
}

bool and_test(){
    bool passed = true;

    bv a = {0xFFFF0000, 0xFFFF0000, 0x0};
    bv b = {0x55550000, 0x55550000, 0x0};

    bv c = {0x5555BBBB, 0x5555BBBB, 0x1};
    bv d = {0xBBBB5555, 0xBBBB5555, 0x1};
    bv e = {0x11111111, 0x11111111, 0x1};

    TEST(((a & c) == b))
    TEST(((c & d) == e))
    TEST(((bv::ones() & bv::zeros()) == bv::zeros()))

    return passed;
}

bool not_test(){
    bool passed = true;

    bv a = {0xFFFF0000, 0xFFFF0000, 0x0};
    bv b = {0x0000FFFF, 0x0000FFFF, 0x1};

    TEST(~bv::ones() == bv::zeros())
    TEST(~a == b)

    return passed;
}

bool has_one_at_test(){
    bool passed = true;

    for(int i = 0; i < bv::bits; i++){
        TEST(bv::ones().has_one_at(i));
    }

    for(int i = 0; i < bv::bits; i++){
        TEST(!bv::zeros().has_one_at(i));
    }

    bv a = {0x11111111, 0x11111111, 0x1};
    for(int i = 0; i < bv::bits; i++){
        TEST(a.has_one_at(i) == (i%4==0))
    }

    return passed;
}

bool single_one_at_test(){
    bool passed = true;

    TEST(bv::single_one_at(0) == bv{0x1})
    TEST(bv::single_one_at(1) == bv{0x2})
    TEST(bv::single_one_at(2) == bv{0x4})
    TEST((bv::single_one_at(34) == bv{0x0, 0x4}))
    TEST((bv::single_one_at(bv::bits-1) == bv{0x0, 0x0, 0x1}))

    return passed;
}

#define BITVECTOR_BITS 64
#define BITVECTOR bv64
#include "bitvector.hpp"
bool insert_bits_test(){
    bool passed = true;

    bv64 a = bv64::zeros();
    a.insert_bits(32, 0xFF);
    a.insert_bits( 0, 0xAA);
    TEST(a == bv64{0xFF000000AA})

    return passed;
}

bool bitvector_tests(){
    bool passed = true;

    passed &= shift_test();
    passed &= or_test();
    passed &= and_test();
    passed &= not_test();
    passed &= has_one_at_test();
    passed &= single_one_at_test();
    passed &= insert_bits_test();

    if(passed){
        cout << "PASSED bitvector tests" << endl;
    }
    return passed;
}
